#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <iostream>

int main() {
    std::cout << "=== CUDA Debugging Information ===" << std::endl;
    
    // 1. Check device count
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    std::cout << "hipGetDeviceCount result: " << hipGetErrorString(error) << std::endl;
    std::cout << "Device count: " << deviceCount << std::endl;
    
    if (error != hipSuccess) {
        return 1;
    }
    
    // 2. Try to set device
    error = hipSetDevice(0);
    std::cout << "cudaSetDevice(0) result: " << hipGetErrorString(error) << std::endl;
    
    // 3. Try to allocate minimal memory
    void* ptr;
    error = hipMalloc(&ptr, 4);
    std::cout << "hipMalloc result: " << hipGetErrorString(error) << std::endl;
    
    if (error == hipSuccess) {
        hipFree(ptr);
        std::cout << "Memory allocation successful!" << std::endl;
    }
    
    // 4. Try to create cuBLAS handle
    hipblasLtHandle_t handle;
    hipblasStatus_t status = hipblasLtCreate(&handle);
    std::cout << "hipblasLtCreate result: " << status << std::endl;
    
    if (status == HIPBLAS_STATUS_SUCCESS) {
        std::cout << "cuBLAS handle created successfully!" << std::endl;
        hipblasLtDestroy(handle);
    }
    
    return 0;
}
