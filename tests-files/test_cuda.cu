#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    
    std::cout << "Found " << deviceCount << " CUDA device(s)" << std::endl;
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device " << i << ": " << prop.name << std::endl;
    }
    
    return 0;
}
